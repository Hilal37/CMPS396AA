#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"

#define THRESHOLD 0.000001
#define YMAX 32
#define BLOCK_DIM 16
#define TILE_DIM 16

__global__ void spmspm(COOMatrix *result, CSRMatrix *A, CSCMatrix *B, float bias) {
    
    //Optimization 2: shared memory tiling

    //the shared arrays
    __shared__ int A_colIdxs_s[TILE_DIM][TILE_DIM];
    __shared__ float A_values_s[TILE_DIM][TILE_DIM];
    __shared__ int B_rowIdxs_s[TILE_DIM][TILE_DIM];
    __shared__ float B_values_s[TILE_DIM][TILE_DIM];

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int rowStartA, rowEndA, colStartB, colEndB, nnzRowA, nnzColB;

    //had to divide if/else like this, to avoid using __syncthreads() inside an if block (=>avoid deadlock)
    if(row < A->numRows && col < B->numCols) {
        rowStartA = A->rowPtrs[row];
        colStartB = B->colPtrs[col];
        rowEndA = A->rowPtrs[row + 1];
        colEndB = B->colPtrs[col + 1];
        nnzRowA = rowEndA - rowStartA;
        nnzColB = colEndB - colStartB;
    }
    else {
        rowStartA = -1;
        colStartB = -1;
        rowEndA = -1;
        colEndB = -1;
        nnzRowA = -1;
        nnzColB = -1;
    }

    float sum = 0.0f;

    for(unsigned int tile = 0; tile < (A->numRows + TILE_DIM - 1)/TILE_DIM; ++tile) {
        //fill shared memory arrays
        unsigned int tileIdx_A = rowStartA + tile*TILE_DIM + threadIdx.x;
        unsigned int tileIdx_B = colStartB + tile*TILE_DIM + threadIdx.y;

        A_colIdxs_s[threadIdx.x][threadIdx.y] = (tileIdx_A < rowEndA) ? A->colIdxs[tileIdx_A] : 0.0f;
        A_values_s[threadIdx.x][threadIdx.y] = (tileIdx_A < rowEndA) ? A->values[tileIdx_A] : 0.0f;
        B_rowIdxs_s[threadIdx.x][threadIdx.y] = (tileIdx_B < colEndB) ? B->rowIdxs[tileIdx_B] : 0.0f;
        B_values_s[threadIdx.x][threadIdx.y] = (tileIdx_B < colEndB) ? B->values[tileIdx_B] : 0.0f;
        __syncthreads();

        //compute (partial) sum for the tile
        unsigned int ia = 0;
        unsigned int ib = 0;
        while(ia < TILE_DIM && ia < nnzRowA && ib < TILE_DIM && ib < nnzColB) {
            unsigned int idxA = A_colIdxs_s[threadIdx.x][ia];
            unsigned int idxB = B_rowIdxs_s[ib][threadIdx.y];
            if(idxA == idxB) {
                sum += A_values_s[threadIdx.x][ia] * B_values_s[ib][threadIdx.y];
                ia++;
                ib++;
            }
            else if(idxA < idxB) {
                ia++;
            }
            else {
                ib++;
            }
        }
        __syncthreads();
    }

    if(sum > THRESHOLD || sum < -THRESHOLD) {
        sum += bias;
        //Remove negative and zero values
        if(sum > 0) {
            if(sum>YMAX) {
                sum = YMAX;
            }
            int nnzIdx = atomicAdd(&result->nnz, 1);
            result->rowIdxs[nnzIdx] = row;
            result->colIdxs[nnzIdx] = col;
            result->values[nnzIdx] = sum;
        }    
    }

}

void findNonzeroRows(Vector* v, CSRMatrix* A) {
    unsigned int nnz = 0;
    for(unsigned int r = 0; r < A->numRows; ++r) {
        unsigned int rowPtrA = A->rowPtrs[r];
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
        if(nnzA > 0) {
            if(nnz >= v->capacity) {
                expandVectorCapacity(v, 2*v->capacity);
            }
            v->data[nnz] = r;
            ++nnz;
        }
    }
    v->nnz = nnz;
}

COOMatrix* createEmptyCOO_d(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
    COOMatrix cooShadow;
    cooShadow.numRows = numRows;
    cooShadow.numCols = numCols;
    cooShadow.nnz = 0;
    cooShadow.capacity = capacity;
    hipMalloc((void**) &cooShadow.rowIdxs, capacity*sizeof(unsigned int));
    hipMalloc((void**) &cooShadow.colIdxs, capacity*sizeof(unsigned int));
    hipMalloc((void**) &cooShadow.values, capacity*sizeof(float));
    COOMatrix* coo_d;
    hipMalloc((void**) &coo_d, sizeof(COOMatrix));
    hipMemcpy(coo_d, &cooShadow, sizeof(COOMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return coo_d;
}

void copyCOOfromGPU(COOMatrix* coo_d, COOMatrix* coo) {
    COOMatrix cooShadow;
    hipMemcpy(&cooShadow, coo_d, sizeof(COOMatrix), hipMemcpyDeviceToHost);
    assert(coo->numRows == cooShadow.numRows);
    assert(coo->numCols == cooShadow.numCols);
    assert(coo->capacity >= cooShadow.nnz);
    coo->nnz = cooShadow.nnz;
    hipMemcpy(coo->rowIdxs, cooShadow.rowIdxs, cooShadow.nnz*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(coo->colIdxs, cooShadow.colIdxs, cooShadow.nnz*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(coo->values, cooShadow.values, cooShadow.nnz*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

CSRMatrix* createEmptyCSR_d(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
    CSRMatrix csrShadow;
    csrShadow.numRows = numRows;
    csrShadow.numCols = numCols;
    csrShadow.nnz = 0;
    csrShadow.capacity = capacity;
    hipMalloc((void**) &csrShadow.rowPtrs, (numRows + 1)*sizeof(unsigned int));
    hipMalloc((void**) &csrShadow.colIdxs, capacity*sizeof(unsigned int));
    hipMalloc((void**) &csrShadow.values, capacity*sizeof(float));
    CSRMatrix* csr_d;
    hipMalloc((void**) &csr_d, sizeof(CSRMatrix));
    hipMemcpy(csr_d, &csrShadow, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return csr_d;
}

void copyCSRtoGPU(CSRMatrix* csr, CSRMatrix* csr_d) {
    CSRMatrix csrShadow;
    hipMemcpy(&csrShadow, csr_d, sizeof(CSRMatrix), hipMemcpyDeviceToHost);
    assert(csrShadow.numRows == csr->numRows);
    assert(csrShadow.numCols == csr->numCols);
    assert(csrShadow.capacity >= csr->nnz);
    csrShadow.nnz = csr->nnz;
    hipMemcpy(csrShadow.rowPtrs, csr->rowPtrs, (csr->numRows + 1)*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrShadow.colIdxs, csr->colIdxs, csr->nnz*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrShadow.values, csr->values, csr->nnz*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

CSCMatrix* createCSCfromCSC_d(CSCMatrix* csc) {
    CSCMatrix cscShadow;
    cscShadow.numRows = csc->numRows;
    cscShadow.numCols = csc->numCols;
    cscShadow.nnz = csc->nnz;
    cscShadow.capacity = csc->capacity;
    hipMalloc((void**) &cscShadow.colPtrs, (csc->numCols + 1)*sizeof(unsigned int));
    hipMalloc((void**) &cscShadow.rowIdxs, csc->capacity*sizeof(unsigned int));
    hipMalloc((void**) &cscShadow.values, csc->capacity*sizeof(float));
    hipMemcpy(cscShadow.colPtrs, csc->colPtrs, (csc->numCols + 1)*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(cscShadow.rowIdxs, csc->rowIdxs, csc->capacity*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(cscShadow.values, csc->values, csc->capacity*sizeof(float), hipMemcpyHostToDevice);
    CSCMatrix* csc_d;
    hipMalloc((void**) &csc_d, sizeof(CSCMatrix));
    hipMemcpy(csc_d, &cscShadow, sizeof(CSCMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return csc_d;
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

    Timer timer;

    // Convert featureVectors to CSR
    startTime(&timer);
    CSRMatrix* Y0 = createEmptyCSR(featureVectors->numRows, featureVectors->numCols, 4*featureVectors->nnz); // Assuming 4*nnz is enough for all Y vectors
    convertCOOtoCSR(featureVectors, Y0);
    CSRMatrix* Y0_d = createEmptyCSR_d(featureVectors->numRows, featureVectors->numCols, 4*featureVectors->nnz); // Assuming 4*nnz is enough for all Y vectors
    stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

    // Convert layer weights to CSC
    startTime(&timer);
    CSCMatrix* W[numLayers];
    CSCMatrix* W_d[numLayers];
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        W[layer] = createCSCfromCOO(layerWeights[layer]);
        W_d[layer] = createCSCfromCSC_d(W[layer]);
    }
    stopTimeAndPrint(&timer, "Convert weights to CSR");

    // Temporary buffer
    startTime(&timer);
    COOMatrix *tmp = createEmptyCOO(Y0->numRows, Y0->numCols, Y0->capacity);
    COOMatrix *tmp_d = createEmptyCOO_d(Y0->numRows, Y0->numCols, Y0->capacity);
    stopTimeAndPrint(&timer, "Allocate temporary buffer");

    // Loop over layers
    CSRMatrix *Yin = Y0;
    COOMatrix *Yout = tmp;
    CSRMatrix *Yin_d = Y0_d;
    COOMatrix *Yout_d = tmp_d;
    for(unsigned int layer = 0; layer < numLayers; ++layer) {

        printf("Computing layer %u (SpMSpM)\n", layer);

        // Copy to GPU
        startTime(&timer);
        copyCSRtoGPU(Yin, Yin_d);
        hipMemset(&Yout_d->nnz, 0, sizeof(unsigned int));
        stopTimeAndPrint(&timer, "    Copy CSR to GPU and clear COO");

        // SpMSpM
        startTime(&timer);
        // TODO: spmspm <<< ..., ... >>> (Yout_d, Yin_d, W_d[layer], bias);
        dim3 numThreadsPerBlock(BLOCK_DIM, BLOCK_DIM);         
	    dim3 numBlocks((Yin->numRows + BLOCK_DIM - 1)/BLOCK_DIM, (W[layer]->numCols + BLOCK_DIM - 1)/BLOCK_DIM);
        spmspm <<< numBlocks, numThreadsPerBlock >>> (Yout_d, Yin_d, W_d[layer], bias);
        hipDeviceSynchronize();
        stopTimeAndPrint(&timer, "    SpMSpM");

        // Copy from GPU
        startTime(&timer);
        copyCOOfromGPU(Yout_d, Yout);
        stopTimeAndPrint(&timer, "    Copy COO from GPU");
        printf("    Output matrix number of nonzeros: %d\n", Yout->nnz);

        // Convert COO to CSR
        startTime(&timer);
        convertCOOtoCSR(Yout, Yin);
        stopTimeAndPrint(&timer, "    Converting COO to CSR");

    }

    // Find nonzero rows
    startTime(&timer);
    findNonzeroRows(result, Yin);
    stopTimeAndPrint(&timer, "Find nonzero rows");

    // Free buffers
    startTime(&timer);
    freeCSR(Y0);
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        freeCSC(W[layer]);
    }
    freeCOO(tmp);
    stopTimeAndPrint(&timer, "Deallocate memory");

}

