#include "hip/hip_runtime.h"

/*
 * Optimization 1: Imnproving thread granularity,
 * by using one thread per output element instead of one per output row.
*/

#include <assert.h>
#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"

#define THRESHOLD 0.000001
#define YMAX 32
#define BLOCK_DIM 32

__global__ void spmspm(COOMatrix *result, CSRMatrix *A, CSCMatrix *B, float bias) {
    
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(row < A->numRows && col < B->numCols) {
        unsigned int nnzRowA = A->rowPtrs[row + 1] - A->rowPtrs[row];
        unsigned int nnzColB = B->colPtrs[col + 1] - B->colPtrs[col];
        unsigned int ia = A->rowPtrs[row];
        unsigned int ib = B->colPtrs[col];
        unsigned int rowEndA = A->rowPtrs[row + 1];
        unsigned int colEndB = B->colPtrs[col + 1];
        if(nnzRowA > 0 && nnzColB > 0) {
            float sum = 0.0f;
            while(ia < rowEndA && ib < colEndB) {
                unsigned int idxA = A->colIdxs[ia];
                unsigned int idxB = B->rowIdxs[ib];
                if(idxA == idxB) {
                    sum += A->values[ia] * B->values[ib];
			        ia++;
			        ib++;
                }
                else if(idxA < idxB) {
                    ia++;
                }
                else {
                    ib++;
                }
            }
            if(sum > THRESHOLD || sum < -THRESHOLD) {
                sum += bias;
                //Remove negative and zero values
                if(sum > 0) {
                    if(sum>YMAX) {
                        sum = YMAX;
                    }
                    int nnzIdx = atomicAdd(&result->nnz, 1);
                    result->rowIdxs[nnzIdx] = row;
                    result->colIdxs[nnzIdx] = col;
                    result->values[nnzIdx] = sum;
                }    
            }
        }
    }

}

void findNonzeroRows(Vector* v, CSRMatrix* A) {
    unsigned int nnz = 0;
    for(unsigned int r = 0; r < A->numRows; ++r) {
        unsigned int rowPtrA = A->rowPtrs[r];
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
        if(nnzA > 0) {
            if(nnz >= v->capacity) {
                expandVectorCapacity(v, 2*v->capacity);
            }
            v->data[nnz] = r;
            ++nnz;
        }
    }
    v->nnz = nnz;
}

COOMatrix* createEmptyCOO_d(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
    COOMatrix cooShadow;
    cooShadow.numRows = numRows;
    cooShadow.numCols = numCols;
    cooShadow.nnz = 0;
    cooShadow.capacity = capacity;
    hipMalloc((void**) &cooShadow.rowIdxs, capacity*sizeof(unsigned int));
    hipMalloc((void**) &cooShadow.colIdxs, capacity*sizeof(unsigned int));
    hipMalloc((void**) &cooShadow.values, capacity*sizeof(float));
    COOMatrix* coo_d;
    hipMalloc((void**) &coo_d, sizeof(COOMatrix));
    hipMemcpy(coo_d, &cooShadow, sizeof(COOMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return coo_d;
}

void copyCOOfromGPU(COOMatrix* coo_d, COOMatrix* coo) {
    COOMatrix cooShadow;
    hipMemcpy(&cooShadow, coo_d, sizeof(COOMatrix), hipMemcpyDeviceToHost);
    assert(coo->numRows == cooShadow.numRows);
    assert(coo->numCols == cooShadow.numCols);
    assert(coo->capacity >= cooShadow.nnz);
    coo->nnz = cooShadow.nnz;
    hipMemcpy(coo->rowIdxs, cooShadow.rowIdxs, cooShadow.nnz*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(coo->colIdxs, cooShadow.colIdxs, cooShadow.nnz*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(coo->values, cooShadow.values, cooShadow.nnz*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

CSRMatrix* createEmptyCSR_d(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
    CSRMatrix csrShadow;
    csrShadow.numRows = numRows;
    csrShadow.numCols = numCols;
    csrShadow.nnz = 0;
    csrShadow.capacity = capacity;
    hipMalloc((void**) &csrShadow.rowPtrs, (numRows + 1)*sizeof(unsigned int));
    hipMalloc((void**) &csrShadow.colIdxs, capacity*sizeof(unsigned int));
    hipMalloc((void**) &csrShadow.values, capacity*sizeof(float));
    CSRMatrix* csr_d;
    hipMalloc((void**) &csr_d, sizeof(CSRMatrix));
    hipMemcpy(csr_d, &csrShadow, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return csr_d;
}

void copyCSRtoGPU(CSRMatrix* csr, CSRMatrix* csr_d) {
    CSRMatrix csrShadow;
    hipMemcpy(&csrShadow, csr_d, sizeof(CSRMatrix), hipMemcpyDeviceToHost);
    assert(csrShadow.numRows == csr->numRows);
    assert(csrShadow.numCols == csr->numCols);
    assert(csrShadow.capacity >= csr->nnz);
    csrShadow.nnz = csr->nnz;
    hipMemcpy(csrShadow.rowPtrs, csr->rowPtrs, (csr->numRows + 1)*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrShadow.colIdxs, csr->colIdxs, csr->nnz*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrShadow.values, csr->values, csr->nnz*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

CSCMatrix* createCSCfromCSC_d(CSCMatrix* csc) {
    CSCMatrix cscShadow;
    cscShadow.numRows = csc->numRows;
    cscShadow.numCols = csc->numCols;
    cscShadow.nnz = csc->nnz;
    cscShadow.capacity = csc->capacity;
    hipMalloc((void**) &cscShadow.colPtrs, (csc->numCols + 1)*sizeof(unsigned int));
    hipMalloc((void**) &cscShadow.rowIdxs, csc->capacity*sizeof(unsigned int));
    hipMalloc((void**) &cscShadow.values, csc->capacity*sizeof(float));
    hipMemcpy(cscShadow.colPtrs, csc->colPtrs, (csc->numCols + 1)*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(cscShadow.rowIdxs, csc->rowIdxs, csc->capacity*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(cscShadow.values, csc->values, csc->capacity*sizeof(float), hipMemcpyHostToDevice);
    CSCMatrix* csc_d;
    hipMalloc((void**) &csc_d, sizeof(CSCMatrix));
    hipMemcpy(csc_d, &cscShadow, sizeof(CSCMatrix), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return csc_d;
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

    Timer timer;

    // Convert featureVectors to CSR
    startTime(&timer);
    CSRMatrix* Y0 = createEmptyCSR(featureVectors->numRows, featureVectors->numCols, 4*featureVectors->nnz); // Assuming 4*nnz is enough for all Y vectors
    convertCOOtoCSR(featureVectors, Y0);
    CSRMatrix* Y0_d = createEmptyCSR_d(featureVectors->numRows, featureVectors->numCols, 4*featureVectors->nnz); // Assuming 4*nnz is enough for all Y vectors
    stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

    // Convert layer weights to CSC
    startTime(&timer);
    CSCMatrix* W[numLayers];
    CSCMatrix* W_d[numLayers];
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        W[layer] = createCSCfromCOO(layerWeights[layer]);
        W_d[layer] = createCSCfromCSC_d(W[layer]);
    }
    stopTimeAndPrint(&timer, "Convert weights to CSR");

    // Temporary buffer
    startTime(&timer);
    COOMatrix *tmp = createEmptyCOO(Y0->numRows, Y0->numCols, Y0->capacity);
    COOMatrix *tmp_d = createEmptyCOO_d(Y0->numRows, Y0->numCols, Y0->capacity);
    stopTimeAndPrint(&timer, "Allocate temporary buffer");

    // Loop over layers
    CSRMatrix *Yin = Y0;
    COOMatrix *Yout = tmp;
    CSRMatrix *Yin_d = Y0_d;
    COOMatrix *Yout_d = tmp_d;
    for(unsigned int layer = 0; layer < numLayers; ++layer) {

        printf("Computing layer %u (SpMSpM)\n", layer);

        // Copy to GPU
        startTime(&timer);
        copyCSRtoGPU(Yin, Yin_d);
        hipMemset(&Yout_d->nnz, 0, sizeof(unsigned int));
        stopTimeAndPrint(&timer, "    Copy CSR to GPU and clear COO");

        // SpMSpM
        startTime(&timer);
        // TODO: spmspm <<< ..., ... >>> (Yout_d, Yin_d, W_d[layer], bias);
        dim3 numThreadsPerBlock(BLOCK_DIM, BLOCK_DIM);         
	    dim3 numBlocks((Yin->numRows + BLOCK_DIM - 1)/BLOCK_DIM, (W[layer]->numCols + BLOCK_DIM - 1)/BLOCK_DIM);
        spmspm <<< numBlocks, numThreadsPerBlock >>> (Yout_d, Yin_d, W_d[layer], bias);
        hipDeviceSynchronize();
        stopTimeAndPrint(&timer, "    SpMSpM");

        // Copy from GPU
        startTime(&timer);
        copyCOOfromGPU(Yout_d, Yout);
        stopTimeAndPrint(&timer, "    Copy COO from GPU");
        printf("    Output matrix number of nonzeros: %d\n", Yout->nnz);

        // Convert COO to CSR
        startTime(&timer);
        convertCOOtoCSR(Yout, Yin);
        stopTimeAndPrint(&timer, "    Converting COO to CSR");

    }

    // Find nonzero rows
    startTime(&timer);
    findNonzeroRows(result, Yin);
    stopTimeAndPrint(&timer, "Find nonzero rows");

    // Free buffers
    startTime(&timer);
    freeCSR(Y0);
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        freeCSC(W[layer]);
    }
    freeCOO(tmp);
    stopTimeAndPrint(&timer, "Deallocate memory");

}

