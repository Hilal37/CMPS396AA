#include "hip/hip_runtime.h"

#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "matrix_gpu0.h"
#include "timer.h"

#define THRESHOLD 0.000001
#define YMAX 32

__global__ void spmspm(CSRMatrix *result, CSRMatrix *A, CSCMatrix *B, float bias) {
    unsigned int nnzIdx= 0;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if(r < A->numRows) {
        unsigned int rowPtrA = A->rowPtrs[r];
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
        if(nnzA>0) {
            unsigned int* colIdxsA = A->colIdxs + rowPtrA;
            float* valueA = A->values + rowPtrA;
            int c = blockIdx.x * blockDim.x + threadIdx.x;
            if(c < B->numCols) {
                unsigned int colPtrB = B->colPtrs[c];
                unsigned int nnzB = B->colPtrs[c + 1] - colPtrB;
                if(nnzB>0) {
                    unsigned int* rowIdxsB = B->rowIdxs + colPtrB;
                    float* valueB = B->values + colPtrB;
                    // Loop and find intersection
                    float sum = 0.0f;
                    unsigned int ia = 0, ib = 0;
                    while(ia < nnzA && ib < nnzB) {
                        unsigned int colIdx = colIdxsA[ia];
                        unsigned int rowIdx = rowIdxsB[ib];
                        if(colIdx < rowIdx) {
                            ia++;
                        } else if(colIdx > rowIdx) {
                            ib++;
                        } else {
                            sum += valueA[ia]*valueB[ib];
                            ia++;
                            ib++;
                        }
                    }

                    if(sum > THRESHOLD || sum < -THRESHOLD) {
                        sum += bias;
                        //Remove negative and zero values
                        if(sum > 0) {
                            if(sum>YMAX) {
                                sum = YMAX;
                            }
                            if(nnzIdx >= result->capacity) {
                                expandCSRCapacity(result, 2*result->capacity);
                            }
                            result->colIdxs[nnzIdx] = c;
                            result->values[nnzIdx] = sum;
                            ++nnzIdx;
                        }    
                    }
                }
            }
        }
        result->rowPtrs[r + 1] = nnzIdx;
    }
    result->nnz = nnzIdx;
}

void findNonzeroRows(Vector* v, CSRMatrix* A) {
    unsigned int nnz = 0;
    for(unsigned int r = 0; r < A->numRows; ++r) {
        unsigned int rowPtrA = A->rowPtrs[r];
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
        if(nnzA > 0) {
            if(nnz >= v->capacity) {
                expandVectorCapacity(v, 2*v->capacity);
            }
            v->data[nnz] = r;
            ++nnz;
        }
    }
    v->nnz = nnz;
}

CSCMatrix* copyCSCToGPU(CSCMatrix* csc) {
    CSCMatrix* csc_d;
    hipMalloc((void **)&csc_d , sizeof(CSRMatrix));
    hipMemcpy(csc_d, csc, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    hipMalloc((void **)&csc_d->colPtrs , (csc->numCols + 1) * sizeof(unsigned int));
    hipMemcpy(csc_d->colPtrs, csc->colPtrs, (csc->numCols + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemset(csc_d->colPtrs, 0, (csc->numCols + 1) * sizeof(unsigned int));
    hipMalloc((void **)&csc_d->rowIdxs , csc->nnz * sizeof(unsigned int));
    hipMemcpy(csc_d->rowIdxs, csc->rowIdxs, csc->nnz * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMalloc((void **)&csc_d->values , csc->nnz * sizeof(float));
    hipMemcpy(csc_d->values, csc->values, csc->nnz * sizeof(float), hipMemcpyHostToDevice);
}

CSCMatrix* copyCSRToGPU(CSRMatrix* csr) {
    CSCMatrix* csr_d;
    hipMalloc((void **)&csr_d , sizeof(CSRMatrix));
    hipMemcpy(csr_d, csr, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    hipMalloc((void **)&csr_d->rowPtrs , (csr->numRows + 1) * sizeof(unsigned int));
    hipMemcpy(csr_d->rowPtrs, csr->rowPtrs, (csr->numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemset(csr_d->rowPtrs, 0, (csr->numRows + 1) * sizeof(unsigned int));
    hipMalloc((void **)&csr_d->colIdxs , csr->nnz * sizeof(unsigned int));
    hipMemcpy(csr_d->colIdxs, csr->colIdxs, csr->nnz * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMalloc((void **)&csr_d->values , csr->nnz * sizeof(float));
    hipMemcpy(csr_d->values, csr->values, csr->nnz * sizeof(float), hipMemcpyHostToDevice);
}

void freeCSRGPU(CSRMatrix* csr) {
    hipFree(csr->rowPtrs);
    hipFree(csr->colIdxs);
    hipFree(csr->values);
    hipFree(csr);
}

void freeCSCGPU(CSCMatrix* csc) {
    hipFree(csc->colPtrs);
    hipFree(csc->rowIdxs);  
    hipFree(csc->values);
    hipFree(csc);
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

    Timer timer;

    // Convert featureVectors to CSR
    startTime(&timer);
    CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
    CSRMatrix* Y0_d = copyCSRToGPU(Y0);
    stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

    // Convert layer weights to CSC
    startTime(&timer);
    CSCMatrix* W[numLayers];
    CSCMatrix* W_d[numLayers];
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        W[layer] = createCSCfromCOO(layerWeights[layer]);
        W_d[layer] = copyCSCToGPU(W[layer]);
    }
    stopTimeAndPrint(&timer, "Convert weights to CSR");

    // Double buffers
    startTime(&timer);
    CSRMatrix *tmp = createEmptyCSR(Y0->numRows, Y0->numCols, 2*Y0->nnz);
    CSRMatrix *tmp_d = copyCSRToGPU(tmp);
    CSRMatrix *inBuffer  = Y0_d;
    CSRMatrix *outBuffer = tmp_d;
    stopTimeAndPrint(&timer, "Allocate temporary buffer");
        
    // Loop over layers
    for(unsigned int layer = 0; layer < numLayers; ++layer) {

        // SpMSpM
        printf("Computing layer %u (SpMSpM)", layer);
        startTime(&timer);
        spmspm(outBuffer, inBuffer, W_d[layer], bias);
        stopTimeAndPrint(&timer, "");

        // Swap buffers
        CSRMatrix *t = inBuffer;
        inBuffer = outBuffer;
        outBuffer = t;

    }

    // Find nonzero rows
    startTime(&timer);
    findNonzeroRows(result, inBuffer);
    stopTimeAndPrint(&timer, "Find nonzero rows");

    // Free buffers
    startTime(&timer);
    freeCSR(Y0);
    freeCSRGPU(Y0_d);
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        freeCSC(W[layer]);
        freeCSCGPU(W_d[layer]);
    }
    freeCSR(tmp);
    freeCSRGPU(tmp_d);
    stopTimeAndPrint(&timer, "Deallocate memory");

}


//ELL Matrix operations

ELLMatrix* createEmptyELLMatrix(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
    ELLMatrix* matrix = (ELLMatrix*)malloc(sizeof(ELLMatrix));
    matrix->numRows = numRows;
    matrix->numCols = numCols;
    matrix->nnz = 0;
    matrix->rowSize = capacity;
    matrix->colIndices = (int*)malloc(numRows*capacity*sizeof(int));
    matrix->values = (float*)malloc(numRows*capacity*sizeof(float));
    matrix->nnzPerRow = (int*)malloc(numRows*sizeof(int))
}

void ELLMatrixAdd(ELLMatrix* matrix, float element, unsigned int row, unsigned int column) {

    if(row >= numRows || col >= numCols || element == 0.0f) {
        return;
    }

    //expand capacity of the ELLMatrix if needed
    if(matrix->nnzPerRow[row] >= matrix->rowSize) {
        ELLMatrixExpand(matrix, matrix->nnzPerRow[row]);
    }

    //the new element's index in the values array
    //unsigned int idx = row*(matrix->rowSize) + matrix->nnzPerRow[row];

    unsigned int idx = ell->numRows * column + row;

    matrix->colIndices[idx] = column;
    matrix->values[idx] = element;
    matrix->nnz++;
    matrix->nnzPerRow[row]++;

}

void ELLMatrixExpand(ELLMatrix* matrix, unsigned int newRowSize) {
    if(newRowSize <= matrix->rowSize) {
        return;
    }

    matrix->colIndices = (int*)realloc(matrix->colIndices, (matrix->numRows)*newRowSize*sizeof(int));
    matrix->values = (float*)realloc(matrix->values, (matrix->numRows)*newRowSize*sizeof(float));
    matrix->rowSize = newRowSize;
}

void ELLMatrixFree(ELLMatrix* matrix) {
    free(matrix->colIndices);
    free(matrix->values);
    free(matrix->nnzPerRow);
    free(matrix);
}

//convert from ELL to CSR in parallel
//assume output pointer has been allocated by host
__global__ void ELLtoCSR(CSRMatrix* output, ELLMatrix* ell, int* rowPtrs) {

    //indices to process
    unsigned int inIdx = blockIdx.x*blockDim.x + threadIdx.x;

    //only need first thread to set these
    if(inIdx == 0) { 
        output->numRows = ell->numRows;
        output->numCols = ell->numCols;
        output->nnz = ell->nnz;
        output->rowIdxs = rowPtrs;
        //TODO: what abt output->capacity ??
    }

    __syncthreads();

    //the matrix row/col of the current element
    unsigned int outRow = inIdx % ell->numRows;
    unsigned int outCol = (int)(inIdx / ell->numRows);
    
    //only add to CSR if the element is an actual number (not a padding)
    //TODO: may need to increase capacity sometimes (depending on initial allocated space)
    if(outCol < ell->nnzPerRow[outRow]) {
        unsigned int outIdx = rowPtrs[outRow] + outCol;
        output->colPtrs[outIdx] = ell->colIndices[inIdx];
        output->values[outIdx] = ell->values[inIdx];
    }

}

//exclusive scan of the ELL's nnzPerRow array
//must be called before converting ELL to CSR (its output is the `rowPtrs` param of ELLtoCSR function)
__global__ int* ELLGetRowPtrs(ELLMatrix* ell) {
    //TODO
}