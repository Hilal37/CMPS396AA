#include "hip/hip_runtime.h"

#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"

#define THRESHOLD 0.000001
#define YMAX 32
#define BLOCK_DIM 1024

__global__ void spmspm(COOMatrix *result, CSRMatrix *A, CSCMatrix *B, float bias) {
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    if(r < A->numRows) {
        unsigned int rowPtrA = A->rowPtrs[r];
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
        if(nnzA>0) {
            unsigned int* colIdxsA = A->colIdxs + rowPtrA;
            float* valueA = A->values + rowPtrA;
            for(unsigned int c = 0; c < B->numCols; c++) {
                unsigned int colPtrB = B->colPtrs[c];
                unsigned int nnzB = B->colPtrs[c + 1] - colPtrB;
                if(nnzB>0) {
                    unsigned int* rowIdxsB = B->rowIdxs + colPtrB;
                    float* valueB = B->values + colPtrB;
                    // Loop and find intersection
                    float sum = 0.0f;
                    unsigned int ia = 0, ib = 0;
                    while(ia < nnzA && ib < nnzB) {
                        unsigned int colIdx = colIdxsA[ia];
                        unsigned int rowIdx = rowIdxsB[ib];
                        if(colIdx < rowIdx) {
                            ia++;
                        } else if(colIdx > rowIdx) {
                            ib++;
                        } else {
                            sum += valueA[ia]*valueB[ib];
                            ia++;
                            ib++;
                        }
                    }

                    if(sum > THRESHOLD || sum < -THRESHOLD) {
                        sum += bias;
                        //Remove negative and zero values
                        if(sum > 0) {
                            if(sum>YMAX) {
                                sum = YMAX;
                            }
                            int nnzIdx = atomicAdd(&result->nnz, 1);
                            result->rowIdxs[nnzIdx] = r;
                            result->colIdxs[nnzIdx] = c;
                            result->values[nnzIdx] = sum;
                        }    
                    }
                }
            }
        }
    }
}

void findNonzeroRows(Vector* v, CSRMatrix* A) {
    unsigned int nnz = 0;
    for(unsigned int r = 0; r < A->numRows; ++r) {
        unsigned int rowPtrA = A->rowPtrs[r];
        unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
        if(nnzA > 0) {
            if(nnz >= v->capacity) {
                expandVectorCapacity(v, 2*v->capacity);
            }
            v->data[nnz] = r;
            ++nnz;
        }
    }
    v->nnz = nnz;
}

CSCMatrix* copyCSCToGPU(CSCMatrix* csc) {
    CSCMatrix* csc_d;
    hipMalloc((void **)&csc_d , sizeof(CSRMatrix));
    hipMemcpy(csc_d, csc, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    hipMalloc((void **)&csc_d->colPtrs , (csc->numCols + 1) * sizeof(unsigned int));
    hipMemcpy(csc_d->colPtrs, csc->colPtrs, (csc->numCols + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemset(csc_d->colPtrs, 0, (csc->numCols + 1) * sizeof(unsigned int));
    hipMalloc((void **)&csc_d->rowIdxs , csc->nnz * sizeof(unsigned int));
    hipMemcpy(csc_d->rowIdxs, csc->rowIdxs, csc->nnz * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMalloc((void **)&csc_d->values , csc->nnz * sizeof(float));
    hipMemcpy(csc_d->values, csc->values, csc->nnz * sizeof(float), hipMemcpyHostToDevice);
    
    return csc_d;
}

CSRMatrix* copyCSRToGPU(CSRMatrix* csr) {
    CSRMatrix* csr_d;
    hipMalloc((void **)&csr_d , sizeof(CSRMatrix));
    hipMemcpy(csr_d, csr, sizeof(CSRMatrix), hipMemcpyHostToDevice);
    hipMalloc((void **)&csr_d->rowPtrs , (csr->numRows + 1) * sizeof(unsigned int));
    hipMemcpy(csr_d->rowPtrs, csr->rowPtrs, (csr->numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMalloc((void **)&csr_d->colIdxs , csr->nnz * sizeof(unsigned int));
    hipMemcpy(csr_d->colIdxs, csr->colIdxs, csr->nnz * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMalloc((void **)&csr_d->values , csr->nnz * sizeof(float));
    hipMemcpy(csr_d->values, csr->values, csr->nnz * sizeof(float), hipMemcpyHostToDevice);
    
    return csr_d;
}

COOMatrix* copyCOOToGPU(COOMatrix* coo) {
    COOMatrix* coo_d;
    hipMalloc((void **)&coo_d , sizeof(COOMatrix));
    hipMemcpy(coo_d, coo, sizeof(COOMatrix), hipMemcpyHostToDevice);
    hipMalloc((void **)&coo_d->rowIdxs , coo->capacity * sizeof(unsigned int));
    hipMemcpy(coo_d->rowIdxs, coo->rowIdxs, coo->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMalloc((void **)&coo_d->colIdxs , coo->capacity * sizeof(unsigned int));
    hipMemcpy(coo_d->colIdxs, coo->colIdxs, coo->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMalloc((void **)&coo_d->values , coo->capacity * sizeof(float));
    hipMemcpy(coo_d->values, coo->values, coo->capacity * sizeof(float), hipMemcpyHostToDevice);
    
    return coo_d;
}

void freeCSRGPU(CSRMatrix* csr) {
    hipFree(csr->rowPtrs);
    hipFree(csr->colIdxs);
    hipFree(csr->values);
    hipFree(csr);
}

void freeCSCGPU(CSCMatrix* csc) {
    hipFree(csc->colPtrs);
    hipFree(csc->rowIdxs);  
    hipFree(csc->values);
    hipFree(csc);
}

void freeCOOGPU(COOMatrix* coo) {
    hipFree(coo->rowIdxs); 
    hipFree(coo->colIdxs); 
    hipFree(coo->values);
    hipFree(coo);
}

COOMatrix* createEmptyCOO_gpu(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
    COOMatrix* coo = (COOMatrix*) malloc(sizeof(COOMatrix));
    coo->numRows = numRows;
    coo->numCols = numCols;
    coo->nnz = 0;
    coo->capacity = capacity;
    coo->rowIdxs = (unsigned int *)malloc(capacity*sizeof(unsigned int));
    coo->colIdxs = (unsigned int *)malloc(capacity*sizeof(unsigned int));
    coo->values = (float *)malloc(capacity*sizeof(float));
    COOMatrix *coo_d = copyCOOToGPU(coo);
    freeCOO(coo);
    
    return coo_d;
}


//custom code: create CSR form COO in parallel

//parallel function for binning
__global__ void binning_kernel(unsigned int* rowPtrs, unsigned int* out_colIdxs, float* out_values, unsigned int* in_rowIdxs, unsigned int* in_colIdxs, float* in_values, unsigned int size) {
    //filling the CSR's colIdxs and values (out_colIdxs and out_values) using the temp rowPtrs
    //NOTE: here rowPtrs is just a temp copy of the actual rowPtrs, so NO NEED TO RESTORE PTRS

    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    unsigned int row = in_rowIdxs[idx];
    unsigned int i = rowPtrs[row]++;


    out_colIdxs[i] = in_colIdxs[idx];
    out_values[i] = in_values[idx];
}

//parallel function to perform scan
__global__ void scan_kernel(unsigned int *input, unsigned int *output) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    __shared__ unsigned int buffer1_s[BLOCK_DIM]; 
    __shared__ unsigned int buffer2_s[BLOCK_DIM]; 
    unsigned int* inBuffer_s = buffer1_s;
    unsigned int* outBuffer_s = buffer2_s; 
    
    if(threadIdx.x == 0) {
        inBuffer_s[threadIdx.x] = 0.0f; } 
    else {
        inBuffer_s[threadIdx.x] = input[i - 1]; 
    }

    __syncthreads();

    for(unsigned int stride = 1; stride <= BLOCK_DIM/2; stride *= 2) { 
        if(threadIdx.x >= stride) {
            outBuffer_s[threadIdx.x] =
            inBuffer_s[threadIdx.x] + inBuffer_s[threadIdx.x - stride];
        } else {
            outBuffer_s[threadIdx.x] = inBuffer_s[threadIdx.x];
        }
        __syncthreads();
        unsigned int* tmp = inBuffer_s; 
        inBuffer_s = outBuffer_s; 
        outBuffer_s = tmp;
    }
    
    // if(threadIdx.x == BLOCK_DIM - 1) { 
    //     partialSums[blockIdx.x] = inBuffer_s[threadIdx.x];
    // }
    output[i] = inBuffer_s[threadIdx.x];
}

//parallel function to compute histogram (adapted from asst. 6)
__global__ void histogram_kernel(float* values, unsigned int* bins, unsigned int size, unsigned int num_bins) {

    extern __shared__ int private_bins[];

    //initializing private bins to zero
    //(for loop is just in case block dim is smaller than the number of bins, 
    //so each thread in the block has to initialize more than one bin to zero)

    for(unsigned int i = 0; i < (num_bins + blockDim.x - 1)/blockDim.x; ++i) {
        if(i*blockDim.x + threadIdx.x < num_bins) {
            private_bins[i*blockDim.x + threadIdx.x] = 0;
        }
    }

    __syncthreads();

    //updating private bins
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < size) {
        unsigned char b = values[idx];
        atomicAdd(&private_bins[b], 1);
    }

    __syncthreads();

    //committing changes to global bins
    //for loop: same logic as the initialization to zero
    for(unsigned int i = 0; i < (num_bins + blockDim.x - 1)/blockDim.x; ++i) {
        if(i*blockDim.x + threadIdx.x < num_bins && private_bins[i*blockDim.x + threadIdx.x] != 0) {
            atomicAdd(&bins[i*blockDim.x + threadIdx.x], private_bins[i*blockDim.x + threadIdx.x]);
        }
    }
}

CSRMatrix* createCSRfromCOO_gpu(COOMatrix* A) {
    //step 1: allocate arrays
    //output arrays
    unsigned int* rowPtrs, *colIdxs;
    float* values;

    hipMalloc((void**) &rowPtrs, (A->numRows + 1) * sizeof(unsigned int));
    hipMemset(rowPtrs, 0, (A->numRows + 1) * sizeof(unsigned int)); //initialize all rowPtrs to zero
    hipMalloc((void**) &colIdxs, A->nnz * sizeof(unsigned int));
    hipMalloc((void**) &values, A->nnz * sizeof(float));

    //copying A->rowIdxs, A->colIdxs, A->values to GPU
    unsigned int* rowIdxs_A, *colIdxs_A;
    float* values_A;
    hipMalloc((void**) &rowIdxs_A, A->nnz*sizeof(unsigned int));
    hipMalloc((void**) &colIdxs_A, A->nnz*sizeof(unsigned int));
    hipMalloc((void**) &values_A, A->nnz*sizeof(float));
    hipMemcpy(rowIdxs_A, A->rowIdxs, A->nnz*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(colIdxs_A, A->colIdxs, A->nnz*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(values_A, A->values, A->nnz*sizeof(float), hipMemcpyHostToDevice);

    //hipDeviceSynchronize();

    //Now we need to compute the rowPtrs (steps 2 and 3)
    //step 2: Histogram (how many non-zeros for each row)
    unsigned int numThreadsPerBlock = 1024;
    unsigned int numBlocks = (A->nnz + numThreadsPerBlock - 1)/numThreadsPerBlock; 
    histogram_kernel <<< numBlocks, numThreadsPerBlock, (A->numRows + 1) * sizeof(int)>>>(values_A, rowPtrs, A->nnz, A->numRows + 1);
    hipDeviceSynchronize();

    //step 3: prefix sum on the rowPtrs
    unsigned int numBlocksScan = (A->numRows + numThreadsPerBlock)/numThreadsPerBlock; //(for the ceiling, + 1 - 1 cancel out)
    scan_kernel<<< numBlocksScan, numThreadsPerBlock >>>(rowPtrs, rowPtrs);
    hipDeviceSynchronize();

    //step 4: binning (populating the colIdxs and values arrays)
    //creating a temp copy of rowPtrs
    unsigned int* rowPtrs_temp;
    hipMalloc(&rowPtrs_temp, (A->numRows + 1)*sizeof(unsigned int));
    hipMemcpy(rowPtrs_temp, rowPtrs, (A->numRows + 1)*sizeof(unsigned int), hipMemcpyDeviceToDevice);

    binning_kernel <<< numBlocks, numThreadsPerBlock >>>(rowPtrs_temp, colIdxs, values, rowIdxs_A, colIdxs_A, values_A, A->nnz);
    hipDeviceSynchronize();

    //copy rowPtrs, colIdxs, and values to CPU (_h for host)
    unsigned int* rowPtrs_h = (unsigned int *) calloc(A->numRows + 1, sizeof(unsigned int));
    unsigned int* colIdxs_h = (unsigned int *) malloc( A->nnz * sizeof(unsigned int));
    float* values_h = (float *)malloc( A->nnz * sizeof(float));

    hipMemcpy(rowPtrs_h, rowPtrs, (A->numRows + 1)*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(colIdxs_h, colIdxs, A->nnz * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(values_h, values, A->nnz * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    //now that we have all the pieces needed, build the CSR and return it
    CSRMatrix* csr = (CSRMatrix*) malloc(sizeof(CSRMatrix));
    csr->numRows = A->numRows;
    csr->numCols = A->numCols;
    csr->nnz = A->nnz;
    csr->capacity = A->nnz;
    csr->rowPtrs = rowPtrs_h;
    csr->colIdxs = colIdxs_h;
    csr->values = values_h;

    return csr;
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

    Timer timer;

    // Convert featureVectors to CSR
    startTime(&timer);
    CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
    CSRMatrix* Y0_d = copyCSRToGPU(Y0);
    stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

    // Convert layer weights to CSC
    startTime(&timer);
    CSCMatrix* W[numLayers];
    CSCMatrix* W_d[numLayers];
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        W[layer] = createCSCfromCOO(layerWeights[layer]);
        W_d[layer] = copyCSCToGPU(W[layer]);
    }
    stopTimeAndPrint(&timer, "Convert weights to CSR");

    // Double buffers
    startTime(&timer);
    CSRMatrix *tmp = createEmptyCSR(Y0->numRows, Y0->numCols, 2*Y0->nnz);
    CSRMatrix *tmp_d = copyCSRToGPU(tmp);
    CSRMatrix *inBuffer  = Y0_d;
    CSRMatrix *outBuffer = tmp_d;
    stopTimeAndPrint(&timer, "Allocate temporary buffer");
    unsigned int numThreadsPerBlock = 1024;
        
    // Loop over layers
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        unsigned int numBlocks = (inBuffer->numRows + numThreadsPerBlock - 1)/numThreadsPerBlock; 

        // SpMSpM
        printf("Computing layer %u (SpMSpM)", layer);
        startTime(&timer);
        COOMatrix* res = createEmptyCOO_gpu(inBuffer->numRows, outBuffer->numCols, inBuffer->numRows * outBuffer->numCols);
        spmspm<<< numBlocks, numThreadsPerBlock >>>(res, inBuffer, W_d[layer], bias);
        stopTimeAndPrint(&timer, "");
        outBuffer = createCSRfromCOO_gpu(res);
        freeCOOGPU(res);
        // Swap buffers
        CSRMatrix *t = inBuffer;
        inBuffer = outBuffer;
        outBuffer = t;

    }

    // Find nonzero rows
    startTime(&timer);
    findNonzeroRows(result, inBuffer);
    stopTimeAndPrint(&timer, "Find nonzero rows");

    // Free buffers
    startTime(&timer);
    freeCSR(Y0);
    freeCSRGPU(Y0_d);
    for(unsigned int layer = 0; layer < numLayers; ++layer) {
        freeCSC(W[layer]);
        freeCSCGPU(W_d[layer]);
    }
    freeCSR(tmp);
    freeCSRGPU(tmp_d);
    stopTimeAndPrint(&timer, "Deallocate memory");

}